
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

// reading and writing a device variable
__device__ int incr = 10;

__global__ void add(int a, int b, int *c)
{
	*c = *c+a+b+incr++;
}

int main(int argc, char **argv)
{
	// test
	int a = 2, b = 3, c;
	int t = 0;
	int *cuda_c = NULL;

	hipMalloc(&cuda_c, sizeof(int));
	hipError_t ret = hipMemcpy(cuda_c, &t, sizeof(int), hipMemcpyHostToDevice);

	sleep(10);
	add<<<1,1>>>(a, b, cuda_c);
	add<<<1,1>>>(a, b, cuda_c);
	hipDeviceSynchronize();
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuda_c);

	printf("%d + %d = %d\n", a, b, c);

	exit(EXIT_SUCCESS);
}
