
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

// reading a UVM global variable
__managed__ int ten = 10;

__global__ void add(int a, int b, int *c)
{
	*c = a+b+ten;
}

int main(int argc, char **argv)
{
	// test
	int a = 2, b = 3, c;
	int *cuda_c = NULL;
	hipMalloc(&cuda_c, sizeof(int));
    sleep(10);
	add<<<1,1>>>(a, b, cuda_c);
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuda_c);

	printf("%d + %d = %d\n", a, b, c);

	exit(EXIT_SUCCESS);
}
