/*
  Source: https://github.com/forresti/stackoverflow_examples/blob/master/testTexcacheObject_1D/testTexcacheObject.cu
*/
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#define N 10

// texture object is a kernel argument
__global__ void printGpu_tex(hipTextureObject_t tex) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N){
        float x = tex1Dfetch<float>(tex, tid);
        printf("tid=%d, tex1Dfetch<float>(tex, %d) = %f \n", tid, tid, x);
    }
}

__global__ void printGpu_vanilla(float* d_buffer) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N){
        float x = d_buffer[tid];
        printf("tid=%d, d_buffer[%d] = %f \n", tid, tid, x);
    }
}

int main() {
    // declare and allocate memory
    float* d_buffer;
    hipMalloc(&d_buffer, N*sizeof(float));

    float h_buffer[10] = {1,2,3,4,5,6,7,8,9,10};
    hipMemcpy(d_buffer, h_buffer, sizeof(float)*N, hipMemcpyHostToDevice);

    //CUDA 5 texture objects: https://developer.nvidia.com/content/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_buffer;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = N*sizeof(float);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    // create texture object: we only have to do this once!
    hipTextureObject_t tex;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);


    //float *d_result;
    //cudaMalloc(&d_result, N*sizeof(float));

    int grid = N/16+1;
    int block = 16;
    printGpu_tex<<<grid, block>>>(tex);
    sleep(10);
    printGpu_vanilla<<<grid, block>>>(d_buffer);


    // destroy texture object
    hipDestroyTextureObject(tex);

    hipFree(d_buffer);
}
