
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
/*
extern "C"
{
 void _ZL24__sti____cudaRegisterAllv(){
 }
} */

__global__ void add(int a, int b, int *c)
{
	*c = a+b;
	printf("Inside %d + %d = %d\n", a, b, *c);
}

int main(int argc, char **argv)
{
	// test
	int a = 3, b = 3, c=0;
	int *cuda_c = NULL;
	printf("before any cuda call\n");
	printf("Hey\n");
	fflush(stdout);
	hipMallocManaged(&cuda_c, sizeof(int));
	add<<<1,1>>>(a, b, cuda_c);
	hipError_t ret = hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	 printf("device 1: %d \n", *cuda_c);

	hipSetDevice(1);
	int *cuda_c2 = NULL;
	hipMallocManaged(&cuda_c2, sizeof(int));
	//cudaSetDevice(0);
	printf("device 1: %d \n", *cuda_c);
	ret = hipMemcpy(cuda_c2, cuda_c, sizeof(int), hipMemcpyDeviceToDevice);
	printf("error: %s \n", hipGetErrorString(ret));	
//        printf("device 1: %d \n", *cuda_c);
//        printf("device 2: %d \n", *cuda_c2);

	hipDeviceSynchronize();
	ret = hipMemcpy(&c, cuda_c2, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//printf("error: %s \n", cudaGetErrorString(ret));
	printf("device 1: %d \n", *cuda_c);
	hipDeviceSynchronize();
	printf("device 2: %d \n", *cuda_c2);
	printf("host %d \n", c);
	fflush(stdout);
	exit(EXIT_SUCCESS);
}
