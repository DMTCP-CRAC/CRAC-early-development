
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

// reading a device variable
__device__ int ten = 10;

__global__ void add(int a, int b, int *c)
{
	*c = a+b+ten;
}

int main(int argc, char **argv)
{
	// test
	int a = 2, b = 3, c = 7;
	int *cuda_c = NULL;

	hipMalloc(&cuda_c, sizeof(int));
	printf (" before sleep \n");
	fflush(stdout);
    sleep(10);
	add<<<1,1>>>(a, b, cuda_c);
	int tmp = 10;
	printf("%d + %d + %d = %d\n", a, b, tmp, c);
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuda_c);

	printf("%d + %d + %d = %d\n", a, b, tmp, c);

	exit(EXIT_SUCCESS);
}
