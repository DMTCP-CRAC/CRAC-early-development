#include "hip/hip_runtime.h"
#include<stdio.h>
#include<unistd.h>
#include<sys/time.h>
#include<stdlib.h>
#include<iostream>

#define BLOCK_SIZE 10

using namespace std;

//----------------------------------- Structures and Globals---------------------------------------------

typedef struct {
	int dimension1;
	int dimension2;	
} ArrayMetadata2D;

// metadata variables describing dimensionalities of all data structures involved in the computation
ArrayMetadata2D A_MD, B_MD, C_MD;
// pointers for input and output arrays in the host memory  
float *A, *B, *C, *C_CPU;
// pointers for input and output arrays in the device memory (NVIDIA DRAM)
float *A_GPU, *B_GPU, *C_GPU;

//----------------------------------- host function definitions -----------------------------------------

void allocateAndInitializeAB();
void computeCpuMMM();
void copyMatricesToGPU();
void copyResultFromGPU();
void compareHostAndGpuOutput();
void die(const char *error); 
void check_error(hipError_t e);

//----------------------------------- CUDA function definitions -----------------------------------------

__global__ void computeGpuMMM(float *A_GPU, float *B_GPU, float *C_GPU, int width);
__global__ void computeGpuMMM_primitive(float *A_GPU, float *B_GPU, float *C_GPU, int width, int B_D2);
extern "C" void __cudaUnregisterFatBinary(void **test);
//-------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {
void *cuda_ptr1 = NULL;
  void *cuda_ptr2 = NULL;	
	A_MD.dimension1 = (argc > 1) ? atoi(argv[1]) : 100;
	A_MD.dimension2 = (argc > 2) ? atoi(argv[2]) : A_MD.dimension1;
	B_MD.dimension1 = (argc > 3) ? atoi(argv[3]) : A_MD.dimension2;
	B_MD.dimension2 = (argc > 4) ? atoi(argv[4]) : B_MD.dimension1;
	int thread_dim_per_block = (argc > 5) ? atoi(argv[5]) : 2;
	C_MD.dimension1 = A_MD.dimension1;
	C_MD.dimension2 = B_MD.dimension2;

	printf("Matrix A is %d-by-%d\n", A_MD.dimension1, A_MD.dimension2);
	printf("Matrix B is %d-by-%d\n", B_MD.dimension1, B_MD.dimension2);
	printf("Matrix C is %d-by-%d\n", C_MD.dimension1, C_MD.dimension2);

	allocateAndInitializeAB();

	//matrix matrix multiplication in the GPU
	dim3 dimGrid(A_MD.dimension1/thread_dim_per_block, B_MD.dimension2/thread_dim_per_block);
	dim3 dimBlock(thread_dim_per_block, thread_dim_per_block);
        copyMatricesToGPU();
	clock_t start = clock();
	computeGpuMMM_primitive<<<dimGrid, dimBlock>>>(A_GPU, B_GPU, C_GPU, A_MD.dimension2, B_MD.dimension2);
	//computeGpuMMM<<<dimGrid, dimBlock>>>(A_GPU, B_GPU, C_GPU, A_MD.dimension2);
	hipDeviceSynchronize();
        clock_t end = clock();
        double elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        printf("Computation time in the GPU: %f seconds\n", elapsed);
        printf("before copying results\n");
	
	copyResultFromGPU();

        printf("after copying results\n");
        // matrix matrix multiplication in the CPU

	sleep(8);
	
	printf("I restarted successefully\n");
	printf("I restarted successefully\n");
	printf("I restarted successefully\n");
       // start = clock();
        computeCpuMMM();
	printf("I I am working just fine\n");
       // end = clock();
       // elapsed = (end - start) / (double) CLOCKS_PER_SEC;
       // printf("Computation time in the CPU: %f seconds\n", elapsed);

	printf("Now I will call cuda malloc and checkpoint here\n");
	hipError_t rc = hipMalloc(&cuda_ptr1, 436*sizeof(char));
	printf("hipMalloc returned: %d, cuda_ptr1: %p\n", (int)rc, cuda_ptr1);
	rc = hipMalloc(&cuda_ptr2, 43*sizeof(char));
	printf("hipMalloc returned: %d, cuda_ptr1: %p\n", (int)rc, cuda_ptr1);
        printf("hipMalloc returned: %d, cuda_ptr2: %p\n", (int)rc, cuda_ptr2);
	computeGpuMMM_primitive<<<dimGrid, dimBlock>>>(A_GPU, B_GPU, C_GPU, A_MD.dimension2, B_MD.dimension2);
//	hipFree(cuda_ptr1);
  //      hipFree(cuda_ptr2);
//	void **bb=NULL;
	//compareHostAndGpuOutput();	
//	__cudaUnregisterFatBinary(bb);
	return 0;
}

// allocate and initialize A and B using a random number generator
void allocateAndInitializeAB() {
	
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	A = (float*) malloc(sizeofA);
	
	srand(time(NULL));
  	for (int i = 0; i < A_MD.dimension1; i++) {
		for (int j = 0; j < A_MD.dimension2; j++) {
			int index = i * A_MD.dimension2 + j;
			A[index] = (rand() % 1000) * 0.001; 
		}
	}
	
	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	B = (float*) malloc(sizeofB);
  	for (int i = 0; i < B_MD.dimension1; i++) {
		for (int j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			B[index] = (rand() % 1000) * 0.001; 
		}
	}
}

// allocate memory in the GPU for all matrices, and copy A and B content from the host CPU memory to the GPU memory
void copyMatricesToGPU() {
	
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &A_GPU, sizeofA));
	check_error(hipMemcpy(A_GPU, A, sizeofA, hipMemcpyHostToDevice));
	
	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &B_GPU, sizeofB));
	check_error(hipMemcpy(B_GPU, B, sizeofB, hipMemcpyHostToDevice));
	
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &C_GPU, sizeofC));
}

// copy results from C_GPU which is in GPU card memory to C_CPU which is in the host CPU for result comparison
void copyResultFromGPU() {
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C_CPU = (float*) malloc(sizeofC);
	check_error(hipMemcpy(C_CPU, C_GPU, sizeofC, hipMemcpyDeviceToHost));
}

// do a straightforward matrix-matrix multiplication in the CPU
// notice that this implementation can be massively improved in the CPU by doing proper cache blocking but we are
// not providing you the efficient CPU implementation as that reveals too much about the ideal GPU implementation
void computeCpuMMM() {
	
	// allocate the result matrix for the CPU computation
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C = (float*) malloc(sizeofC);
	
	// compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
	for (int i = 0; i < A_MD.dimension1; i++) {
		int a_i = i * A_MD.dimension2;
		int c_i = i * C_MD.dimension2;
		for (int j = 0; j < B_MD.dimension2; j++) {
			int c_index = c_i + j;
			C[c_index] = 0;
			for (int k = 0; k < B_MD.dimension1; k++) {
				int a_index = a_i + k;
				int b_index = k * B_MD.dimension2 + j;
				C[c_index] += A[a_index] * B[b_index];
			}
		}
	}
}

__global__ void computeGpuMMM(float *A_GPU, float *B_GPU, float *C_GPU, int width){
	
	// getting position index of the thread in device
	int t_x = threadIdx.x, t_y = threadIdx.y;
	int b_x = blockIdx.x,  b_y = blockIdx.y;
	int row = b_y*blockDim.y + t_y;
	int col = b_x*blockDim.x + t_x;
	
	// allocate shared memory in block for threads
	__shared__ float s_a[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float s_b[BLOCK_SIZE][BLOCK_SIZE];

	float result = 0;

  	//In the unit of Block in device, computing the Blocking that each unit is in charge of by
	//dot product-ing each Blocking in A & B
  	for(int b = 0; b < width/BLOCK_SIZE; b++)
  	{
    		// load Blocking into shared memory, cooperated by all the threads in the Block in device
    		s_a[t_y][t_x] = A_GPU[row*width + (b*BLOCK_SIZE + t_x)];
    		s_b[t_y][t_x] = B_GPU[(b*BLOCK_SIZE + t_y)*width + col];
    		__syncthreads();
    		
		// dot product on current loaded Blocking
    		for(int i = 0; i < BLOCK_SIZE; i++)
      			result += s_a[t_y][i] * s_b[i][t_x];
    		__syncthreads();
  	}
	
	C_GPU[row*width+col] = result;
}

__global__ void computeGpuMMM_primitive(float *A_GPU, float *B_GPU, float *C_GPU, int width, int B_D2){

        // getting position index of the thread in device
        int row = blockIdx.y*blockDim.y + threadIdx.y;
        int col = blockIdx.x*blockDim.x + threadIdx.x;

        float result = 0;

        // dot product
        for(int i = 0; i < width; i++)
                result += A_GPU[row*width+i] * B_GPU[i*B_D2+col];

        C_GPU[row*B_D2+col] = result;
}

// function to determine if the GPU computation is done correctly by comparing the output from the GPU with that
// from the CPU
void compareHostAndGpuOutput() {
	int totalElements = C_MD.dimension1 * C_MD.dimension2;
	int missmatchCount = 0;
	for (int i = 0; i < totalElements; i++) {
		if (fabs(C[i] - C_CPU[i]) > 0.01) {
			missmatchCount++;
			printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
		}
		else{
			printf("match at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
		}
	}
	if (missmatchCount > 0) {
		printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
	} else {
		printf("Computation is correct: CPU and GPU outputs match\n");
	}
}

// Prints the specified error message and then exits
void die(const char *error) {
        printf("%s", error);
        exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
        if (e != hipSuccess) {
                printf("\nCUDA error: %s\n", hipGetErrorString(e));
                exit(1);
        }
}

