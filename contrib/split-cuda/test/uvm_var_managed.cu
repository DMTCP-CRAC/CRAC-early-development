
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

// using a UVM global variable (cudaMallocManaged) that is set first on the device

__global__ void set_ten(int *ten)
{
	*ten = 10;
}

__global__ void add(int a, int b, int *c, int *ten)
{
	*c = a+b+*ten;
}

int main(int argc, char **argv)
{
	// test
	int a = 2, b = 3, c;
	int *cuda_c = NULL, *ten = NULL;

	hipMalloc(&cuda_c, sizeof(int));
	hipMallocManaged(&ten, sizeof(int));
    sleep(10);
	set_ten<<<1,1>>>(ten);
	add<<<1,1>>>(a, b, cuda_c, ten);
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuda_c);

	printf("%d + %d + %d= %d\n", a, b, *ten, c);

	exit(EXIT_SUCCESS);
}
